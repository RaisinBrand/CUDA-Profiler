#include <stdio.h>
#include <hip/hip_runtime.h>


/* Simple C macro to check CUDA errors */
#define CHECK_CUDA(call)                                       \
    do {                                                       \
        hipError_t err = (call);                              \
        if (err != hipSuccess) {                              \
            fprintf(stderr,                                    \
                "CUDA error at %s:%d: %s\n",                   \
                __FILE__, __LINE__, hipGetErrorString(err));  \
            exit(EXIT_FAILURE);                                \
        }                                                      \
    } while (0)

__global__
void vectorAdd(int *A, int *B, int *C, int numElements){

    //compute the threads global index for 1D grid of 1D rows essentially [block1, block2, ...]
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    
    if(i < numElements){ //prevent out of bound access
        C[i] = A[i] + B[i]; //each thread will do its own globally indexed addition
    }
    
}

int main(){
    int numElements = 8;
    //create pointer A that points to the start of an integer array that is numElements size
    int *A = (int*)malloc(numElements *sizeof(int));
    int *B = (int*)malloc(numElements *sizeof(int));
    int *C = (int*)malloc(numElements *sizeof(int));
    if(A == NULL || B == NULL || C == NULL){
        printf("Memory allocation failed\n");
        exit(0);
    }

    for(int i = 0; i < numElements; i++){
        A[i] = i;
        B[i] = i * 2;
    }

    for (int i = 0; i < numElements; i++){
        printf("A[%d] = %d B[%d] = %d \n", i, A[i], i, B[i]);
    }

    free(A);
    free(B);
    free(C);
}